
#include <hip/hip_runtime.h>
#include <memory>
#include <vector>
#include <string>
#include <cstdint>
#include <fstream>
#include <sstream>
#include <iostream>
#include <getopt.h>
#include <functional>
#include <hip/hip_runtime_api.h>


// Set to 0 to disable debugging
#define DDEBUG 1
#define MAX_GPU_THREADS 256

// CUDA Helper Directives
#define checkCudaErrors(call)                                           \
    do {                                                                \
        hipError_t err = call;                                         \
        if (err != hipSuccess) {                                       \
            std::cerr << "CUDA error at " << __FILE__ << " "            \
                      << __LINE__ << ": " << hipGetErrorString(err)    \
                      << std::endl;                                     \
            std::exit(EXIT_FAILURE);                                    \
        }                                                               \
    } while (0)

typedef struct {
    std::vector<std::int32_t> left;
    std::vector<std::int32_t> right;
} InputData;

// Performs final bitonic check and swaps if conditions aren't meant
__device__ void checkAndSwap(int32_t* arr, int32_t current_position, int32_t other_position) {
    // Store in registers to avoid multiple global reads
    int32_t current = arr[current_position], other = arr[other_position];
    // Perform the swap
    if (other < current) {
        arr[current_position] = other;
        arr[other_position] = current;
    }
}

// Generates bitonic sequences within the array (note: we do this for both arrays)
__global__ void sortIntoBitonicSequences(int32_t* left, int32_t* right, size_t size, size_t mask) {
    size_t current_position = blockIdx.x * blockDim.x + threadIdx.x;
    if (current_position < size) {
        size_t other_position = current_position ^ mask;
        if (current_position < other_position && other_position < size) {
            checkAndSwap(left, current_position, other_position);
            checkAndSwap(right, current_position, other_position);
        }
    }
}

// Does a parallel reduction by taking different of two arrays at a given position
__global__ void differenceReduction(int32_t* left, int32_t* right, int32_t* answer, size_t size) {
    extern __shared__ int32_t shared_memory[];
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t threadIndex = threadIdx.x;

    // Store in shared memory (less reads to global memory)
    if (index < size) {
        shared_memory[threadIndex] = abs(left[index] - right[index]);
    } else {
        shared_memory[threadIndex] = 0;
    }
    __syncthreads();

    // Perform parallel reduction (similarly to a tree converging to the root)
    for (size_t stride = 1; stride < blockDim.x; stride = stride << 1) {
        if ((threadIndex % (stride << 1)) == 0) {
            shared_memory[threadIndex] += shared_memory[threadIndex + stride];
        }
        __syncthreads();
    }

    // Write result atomically to global memory
    if (threadIndex == 0) {
        atomicAdd(answer, shared_memory[threadIndex]);
    }

}

__global__ void findFrequencies(int32_t* frequencies, int32_t* right, int32_t size) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        atomicAdd(&frequencies[right[index]], 1);
    }

}

// Does a parallel reduction for similarlity score
__global__ void similarityScoreReduction(int32_t* frequencies, int32_t* left, int32_t* answer, size_t size) {
    extern __shared__ int32_t shared_memory[];
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t threadIndex = threadIdx.x;

    // Store in shared memory (less reads to global memory)
    if (index < size) {
        shared_memory[threadIndex] = left[index] * frequencies[left[index] ];
    } else {
        shared_memory[threadIndex] = 0;
    }
    __syncthreads();

    // Perform parallel reduction (similarly to a tree converging to the root)
    for (size_t stride = 1; stride < blockDim.x; stride = stride << 1) {
        if ((threadIndex % (stride << 1)) == 0) {
            shared_memory[threadIndex] += shared_memory[threadIndex + stride];
        }
        __syncthreads();
    }

    // Write result atomically to global memory
    if (threadIndex == 0) {
        atomicAdd(answer, shared_memory[threadIndex]);
    }

}

void solvePartOne(std::unique_ptr<InputData>& data) {
    if (data->left.size() != data->right.size()) {
        std::cerr << "Error: Sizes don't match for left and right array" << std::endl;
    }

    std::int32_t* host_answer = nullptr;
    std::int32_t* device_answer = nullptr;
    std::int32_t* device_left = nullptr;
    std::int32_t* device_right = nullptr;
    size_t size = data->left.size();
    std::size_t nThreads = MAX_GPU_THREADS;
    std::size_t nBlocks = (size + nThreads - 1) / nThreads;

    // Allocate pinned host memory
    checkCudaErrors(hipHostAlloc(&host_answer, sizeof(std::int32_t), hipHostMallocDefault));

    // Allocate device memory and copy over the unsorted array to device
    checkCudaErrors(hipMalloc(&device_answer, sizeof(std::int32_t)));
    checkCudaErrors(hipMalloc(&device_left, sizeof(std::int32_t) * size));
    checkCudaErrors(hipMalloc(&device_right, sizeof(std::int32_t) * size));
    checkCudaErrors(hipMemcpy(device_left, data->left.data(), sizeof(std::int32_t) * size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(device_right, data->right.data(), sizeof(std::int32_t) * size, hipMemcpyHostToDevice));

    // This is moditification of biotonic sort to work on non-powers of 2
    // k is the size of the bitonic sequence (starting with base case solved k = 1)
    // It will iterate from size 2, 4, 8, 16....
    // Below is an example (i: increasing, d: decreasing, >: start):
    // > [ 2 9 4 1 6 3 0 5 ]
    //   [ 2 9 4 1 3 6 5 0 ]
    //     i   d   i   d
    //   [ 1 2 4 9 6 5 3 0 ]
    //     i       d
    //   [ 0 1 2 3 4 5 6 9 ]
    //     i
    for (std::size_t k = 2; (k >> 1) < size; k = k << 1) {
        // Deals with the elements that don't fall into the power of 2 case (forward comparator)
        sortIntoBitonicSequences<<<nThreads, nBlocks>>>(device_left, device_right, size, k - 1);
        // Main forward comparator
        for (std::size_t j = k >> 1; j > 0; j = j >> 1) {
            sortIntoBitonicSequences<<<nThreads, nBlocks>>>(device_left, device_right, size, j);
        }
    }

    // Performs a parallel reduction (this is done considering the different between left and right list)
    differenceReduction<<<nThreads, nBlocks, MAX_GPU_THREADS * sizeof(std::int32_t)>>>(device_left, device_right, device_answer, size);

    // Copy back the answer from GPU to CPU
    checkCudaErrors(hipMemcpy(host_answer, device_answer, sizeof(std::int32_t), hipMemcpyDeviceToHost));
    std::cout << "Part 1 Answer: " << *host_answer << std::endl;

    // Copy the sorted data back to the host and free device memory
    checkCudaErrors(hipFree(device_answer));
    checkCudaErrors(hipFree(device_left));
    checkCudaErrors(hipFree(device_right));
}

void solvePartTwo(std::unique_ptr<InputData>& data) {
    if (data->left.size() != data->right.size()) {
        std::cerr << "Error: Sizes don't match for left and right array" << std::endl;
    }

    std::int32_t* host_answer = nullptr;
    std::int32_t* device_answer = nullptr;
    std::int32_t* device_left = nullptr;
    std::int32_t* device_right = nullptr;
    std::int32_t* device_frequencies = nullptr;
    size_t size = data->left.size();
    size_t max_number = 150000;
    std::size_t nThreads = MAX_GPU_THREADS;
    std::size_t nBlocks = (size + nThreads - 1) / nThreads;

    // Allocate pinned host memory
    checkCudaErrors(hipHostAlloc(&host_answer, sizeof(std::int32_t), hipHostMallocDefault));

    // Allocate device memory and copy over the unsorted array to device
    checkCudaErrors(hipMalloc(&device_answer, sizeof(std::int32_t)));
    checkCudaErrors(hipMalloc(&device_left, sizeof(std::int32_t) * size));
    checkCudaErrors(hipMalloc(&device_right, sizeof(std::int32_t) * size));
    checkCudaErrors(hipMalloc(&device_frequencies, sizeof(std::int32_t) * max_number));
    checkCudaErrors(hipMemcpy(device_left, data->left.data(), sizeof(std::int32_t) * size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(device_right, data->right.data(), sizeof(std::int32_t) * size, hipMemcpyHostToDevice));

    // Performs parallel frequency counting
    findFrequencies<<<nThreads, nBlocks>>>(device_frequencies, device_right, size);

    // Performs similarity score calcuating with parallel reduction
    similarityScoreReduction<<<nThreads, nBlocks>>>(device_frequencies, device_left, device_answer, size);

    // Copy back the answer from GPU to CPU
    checkCudaErrors(hipMemcpy(host_answer, device_answer, sizeof(std::int32_t), hipMemcpyDeviceToHost));
    std::cout << "Part 2 Answer: " << *host_answer << std::endl;

    // Copy the sorted data back to the host and free device memory
    checkCudaErrors(hipFree(device_answer));
    checkCudaErrors(hipFree(device_left));
    checkCudaErrors(hipFree(device_right));
    checkCudaErrors(hipFree(device_frequencies));
}

void usage(char* argv[]) {
    std::cout << "Usage: " << argv[0] <<" [OPTIONS]" << std::endl
              << "Options:" << std::endl
              << "  -f, --file <file>    Advent of code sample file" << std::endl
              << "  -debug, --debug      Sets debug flag" << std::endl;

    return;
}

std::string parseArgs(std::int32_t& argc, char* argv[]) {
    struct option longOptions[] = {{"file", required_argument, NULL, 'f'}};
    std::string filename;
    char flag;

    while ((flag = getopt_long(argc, argv, "f:", longOptions, NULL)) != -1) {
        switch (flag) {
            case 'f':
                filename = std::string(optarg);
                break;
            default:
                usage(argv);
        }
    }

    if (filename.empty()) {
        usage(argv);
        exit(EXIT_FAILURE);
    }

    return filename;
}

void parseFile(std::string& filename, 
               std::unique_ptr<InputData>& data, 
               std::function<void(std::istringstream, std::unique_ptr<InputData>&)> func) {
                
    std::ifstream file(filename);
    std::string line;

    if (!file.is_open()) {
        std::cerr << "Error: Couldn't open file" << std::endl;
        std::exit(EXIT_FAILURE);
    }

    while (std::getline(file, line)) {
        func(std::istringstream(line), data);
    }

    return;
}

void dayOneParse(std::istringstream iss, std::unique_ptr<InputData>& data) {
    std::int32_t left_num, right_num;
    iss >> left_num >> right_num;
    data->left.push_back(left_num);
    data->right.push_back(right_num);
    return;
}

int main(std::int32_t argc, char* argv[]) {
    std::unique_ptr<InputData> data = std::make_unique<InputData>();
    std::string filename = parseArgs(argc, argv);
    parseFile(filename, data, dayOneParse);

    // Day 1: Sort and find difference between positions on list
    solvePartOne(data);

    // Day 2: Count frequencies and find similarlity score
    solvePartTwo(data);
}